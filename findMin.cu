#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <iostream>


#define dllexp __declspec(dllexport)
#define byte uint8_t
#define ushort unsigned short

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

extern "C" {
	dllexp void FindMin(uint size);

	__global__ void findMin(ushort* input, uint size, ushort* value) {
		uint tid = blockIdx.x * blockDim.x + threadIdx.x;
		uint half = size / 2;
		byte odd = half % 2;

		while (half > 0) {
			if (tid == 0 && odd == 1) {
				input[tid] = input[tid] > input[half] ? input[half] : input[tid];
				input[tid] = input[tid] > input[half * 2 - 1] ? input[half * 2 - 1] : input[tid];
				*value = input[tid];
			}
			else {
				input[tid] = input[tid] > input[half + tid] ? input[half + tid] : input[tid];
			}
			__syncthreads();

			if (half == 1) {
				half = 0;
			}
			else {
				half /= 2;
				odd = half % 2;
			}
		}
	}

	void FindMin(uint size) {
		ushort* a = new ushort[size];
		ushort minValue = 0;
		ushort* dev_a = new ushort[size];
		ushort* dev_minValue = 0;

		hipMalloc((void**)&dev_a, sizeof(ushort) * size);
		hipMalloc((void**)&dev_minValue, sizeof(ushort));

		srand(time(NULL));
		for (int i = 0; i < size; i++) {
			a[i] = rand() % 256;
			std::cout << a[i] << " " << std::endl;
		}

		hipMemcpy(dev_a, a, sizeof(ushort) * size, hipMemcpyHostToDevice);

		uint threadCount = size / 2;
		printf("threadCount: %i\n", threadCount);
		hipError_t cudaStatus;

		findMin << <1, threadCount >> > (dev_a, size, dev_minValue);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "findMin kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		hipMemcpy(a, dev_a, sizeof(ushort) * size, hipMemcpyDeviceToHost);
		hipMemcpy(&minValue, dev_minValue, sizeof(ushort), hipMemcpyDeviceToHost);

		std::cout << "Min value: " << minValue << std::endl;

		Error:

		hipFree(dev_a);
		hipFree(dev_minValue);
		delete a;
	}
